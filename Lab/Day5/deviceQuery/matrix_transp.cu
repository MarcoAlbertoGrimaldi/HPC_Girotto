
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define N (2048*2048)
#define THREAD_PER_BLOCK 512

__global__ void tra( int *a, int *b) {
        int i = blockIdx.x/4;
        int j = (blockIdx.x%4) * blockDim.x + threadIdx.x;
    
        b[i*2048+j] = a[j*2048+i];
}

void random_ints(int *p, int n) {
	int i;
	for(i=0; i<n; i++) {
		p[i]=rand();
	}
}

int main( void ) {
    int *a, *b, *c;               // host copies of a, b, c
    int *dev_a, *dev_b;   // device copies of a, b, c
    int size = N * sizeof( int ); // we need space for N   									// integers
    int i, j;

    // allocate device copies of a, b
    hipMalloc( (void**)&dev_a, size );
    hipMalloc( (void**)&dev_b, size );

    a = (int*)malloc( size ); 
    b = (int*)malloc( size );
    c = (int*)malloc( size );

    random_ints( a, N ); 
    random_ints( b, N );
    // copy inputs to device
   hipMemcpy( dev_a, a, size, hipMemcpyHostToDevice );
   hipMemcpy( dev_b, b, size, hipMemcpyHostToDevice );

    // launch an rev() kernel with N threads
    tra<<< N/THREAD_PER_BLOCK, THREAD_PER_BLOCK >>>( dev_a, dev_b);

    // copy device result back to host copy of c
   hipMemcpy( b, dev_b, size,   hipMemcpyDeviceToHost );

    for(i=0; i<2048; i++) {
        for(j=0; j<2048; j++) {
            c[i*2048+j] = a[j*2048+i];
            if(b[i*2048+j]!=c[i*2048+j]) {
                printf("error: expected %d, got %d!\n",c[i*2048+j], b[i*2048+j]);
                break;
            }
        }
    }

    if(i==N) {printf("correct!\n");}
 
    free( a ); free( b ); free( c );
    hipFree( dev_a );
    hipFree( dev_b );
    return 0;
}