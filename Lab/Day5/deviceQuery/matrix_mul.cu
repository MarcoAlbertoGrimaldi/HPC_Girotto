#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

#define N (2048*2048)
#define THREAD_PER_BLOCK 512

__global__ void mul( int *a, int *b, int *c) {
        int i = blockIdx.x/4;
        int j = (blockIdx.x%4) * blockDim + threadIdx.x;
        c[i*2048+j]=0;
        for(int k=0; k<N; ++k){
            c[i*2048+j] += a[i*2048+k]*a[k*2048+j];
        }
}

void random_ints(int *p, int n) {
	int i;
	for(i=0; i<n; i++) {
		p[i]=rand();
	}
}

int main( void ) {
    int *a, *b, *c, *d;               // host copies of a, b, c
    int *dev_a, *dev_b, *dev_c;   // device copies of a, b, c
    int size = N * sizeof( int ); // we need space for N   									// integers
    int i;

    // allocate device copies of a, b
    hipMalloc( (void**)&dev_a, size );
    hipMalloc( (void**)&dev_b, size );

    a = (int*)malloc( size ); 
    b = (int*)malloc( size );
    c = (int*)malloc( size );
    d = (int*)malloc( size );

    random_ints( a, N ); 
    random_ints( b, N );

    // copy inputs to device
   hipMemcpy( dev_a, a, size, hipMemcpyHostToDevice );
   hipMemcpy( dev_b, b, size, hipMemcpyHostToDevice );

    // launch an rev() kernel with N threads
    rev<<< N/THREAD_PER_BLOCK, THREAD_PER_BLOCK >>>( dev_a, dev_b);

    // copy device result back to host copy of c
   hipMemcpy( b, dev_b, size,   hipMemcpyDeviceToHost );

    for(i=0; i<N; i++) {
            d[i] += 0;   
    }

    for(i=0; i<2048; i++) {
        for(j=0; j<2048; j++0) {
            for(k=0; k<2048; k++0) {
                d[i*2048+j] += a[i*2048+k]*b[k*2048+j];
                if(b[i]!=d[i]) {
                    printf("error: expected %d, got %d!\n",d[i], c[i]);
                    break;
            }
        }
        if(i==N) {
            printf("correct!\n");
         }  
    }
 
    free( a ); free( b ); free( c ); free( d );
    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_d );
    return 0;
}